#include "Timer.h"

Timer::Timer(){
	hipEventCreate(&begin);
	hipEventCreate(&end);
}

void Timer::start(){
	hipEventRecord(begin);
}

void Timer::stop(){
	hipEventRecord(end);
}

void Timer::printTime(){
	hipEventSynchronize(end);
	float tempo=0;
	hipEventElapsedTime(&tempo, begin, end);
	cout << tempo << endl;//" ms" << endl;
}

float Timer::getTime(){
	hipEventSynchronize(end);
	float tempo=0;
	hipEventElapsedTime(&tempo, begin, end);
	return float(tempo);
}

